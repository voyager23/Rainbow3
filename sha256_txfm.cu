// filename: sha256_txfm.cu

//======================================================================
__device__ __host__
void sha256_transform(uint32_t *w, uint32_t *H);
//======================================================================
__device__ __host__
void sha256_transform(uint32_t *w, uint32_t *H) {
	//working variables 32 bit words
	int i;
	uint32_t a,b,c,d,e,f,g,h,T1,T2;
	// 22Apr2012 k[] array added to function
	uint32_t k[64] = {
	   0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	   0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	   0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	   0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	   0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	   0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	   0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	   0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2 };

	a = H[0];
	b = H[1];
	c = H[2];
	d = H[3];
	e = H[4];
	f = H[5];
	g = H[6];
	h = H[7];
	
	//DEBUG
	//printf("sha256_txfm:\n");
	//for(i=0;i<64;i++) printf("k=%08x w=%08x\n", k[i],w[i]);
	//END DEBUG
	
   for (i = 0; i < 64; ++i) {  
      T1 = h + EP1(e) + CH(e,f,g) + k[i] + w[i];
      T2 = EP0(a) + MAJ(a,b,c);
      h = g;
      g = f;
      f = e;
      e = d + T1;
      d = c;
      c = b;
      b = a;
      a = T1 + T2;
      
      //DEBUG
	  //printf("sha256_txfm:\n");
	  //printf("T1: %08x\n",T1);
	  //printf("T2: %08x\n",T2);
	  //END DEBUG
	  
  }      
    // compute single block hash value
	H[0] += a;
	H[1] += b;
	H[2] += c;
	H[3] += d;
	H[4] += e;
	H[5] += f;
	H[6] += g;
	H[7] += h;
}
//======================================================================
