// filename: freduce.cu


#include <hip/hip_runtime.h>
#include <stdint.h>

//======================================================================
#ifdef __CUDA__ 
__device__ __host__
#endif
void reduce_hash(uint32_t H[], uint8_t B[], int link_idx, uint32_t tab_id);
//======================================================================
#ifdef __CUDA__ 
__device__ __host__
#endif
void reduce_hash(uint32_t H[], uint8_t B[], int link_idx, uint32_t tab_id) {

		uint32_t z;
		uint16_t b0,b1;
		const uint16_t mask = 0xffff;
		
		uint32_t offset = (link_idx+tab_id);
		
		z = H[0] + offset;
		b0 = (uint16_t)(z & mask);
		B[0] = (b0 % 26) + 'A';
		z >>= 16;
		b1 = (uint16_t)(z & mask);
		B[1] = (b1 % 26) + 'A';

		z = H[1] + offset;
		b0 = (uint16_t)(z & mask);
		B[2] = (b0 % 10) + '0';
		z >>= 16;
		b1 = (uint16_t)(z & mask);
		B[3] = (b1 % 10) + '0';

		z = H[2] + offset;
		b0 = (uint16_t)(z & mask);
		B[4] = (b0 % 26) + 'a';
		z >>= 16;
		b1 = (uint16_t)(z & mask);
		B[5] = (b1 % 26) + 'a';
	
		z = H[3] + offset;
		b0 = (uint16_t)(z & mask);
		B[6] = (b0 % 26) + 'A';
		B[7] = '\0';
}
//=============================================================================
