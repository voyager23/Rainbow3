#include "hip/hip_runtime.h"
/*	
	*
	* searchtable_v4.cu
	* 05Apr2012
	* Incorporating new code from the threaded searchtable_v7.c
	* nvcc -Xlinker -lm searchtable_v3.cu table_utils.c md5.c
	* Updated version 22Apr2012
	*
*/

#ifndef __CUDA__
	#define __CUDA__
#endif

//===========================Include headers===============================

#include "rainbow.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <endian.h>
#include <time.h>

//=========================Declarations=================================
__global__
void kernel(TableHeader *header, TableEntry *entry);
//=========================Definitions==================================
__host__
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//Removed hash constant k[64] from here - 23Apr2012

//=========================Include functions and utilities==============
#include "freduce.cu"
#include "initHash.cu"
#include "sha256_txfm.cu"
#include "utils.cu"
//=========================Kernel=======================================
__global__
void kernel(TableHeader *header, TableEntry *entry) {
/*
	* revised 23Apr2012
	* The parameter is the base address of a large table of TableEntry(s)
	* Derived from table_calculate - given a target hash calculate a table
	* of candidate hashes
	* Algorithm takes input_hash and calculates final_hash and sublinks value.
*/

	uint8_t  M[64];	// Initial string - zero padded and length in bits appended
	uint32_t W[64];	// Expanded Key Schedule
	uint32_t H[8];	// Hash
	int i = 0;		// working index
	uint64_t l = 0; // length of message
	uint8_t  B[64];	// store initial and working passwords here to protect original data
	uint8_t *in,*out;
	
	int reduction_idx,count;

	uint thread_idx = blockIdx.x*blockDim.x + threadIdx.x;
	

	if(thread_idx<LINKS) {
	
		// set up a pointer to input_hash & final_hash
		TableEntry *data = entry + thread_idx;
		// move target hash to H
		for(i=0;i<8;i++) H[i] = data->input_hash[i];

		reduction_idx = thread_idx;
		count = LINKS - thread_idx - 1;

		while(count > 0) {
			// Reduce hash to zero terminated password in B
			// Use freduce.cu
			reduce_hash(H,B,reduction_idx);

			// copy zero terminated string from B to M and note length
			in = B;
			out = M;
			i=0; l=0;
			while(in[i] != 0x00) {
				out[i] = in[i];
				i++;
				l++;
			}
			out[i++] = 0x80;
			// zero fill
			while(i < 56) out[i++]=0x00;
			/*
				 * The hash algorithm uses 32 bit (4 byte words).
				 * On little endian machines (Intel) the constants
				 * are stored lsb->msb internally. To match this the WORDS
				 * of the input message are subject to endian swap.
			*/
			uint8_t *x = M;
			int y;
			for(y=0; y<14; y++) {
				// long swap
				*(x+3) ^= *x;
				*x     ^= *(x+3);
				*(x+3) ^= *x;
				// short swap
				*(x+2) ^= *(x+1);
				*(x+1) ^= *(x+2);
				*(x+2) ^= *(x+1);
				// move pointer up
				x += 4;
			}
			// need a 32 bit pointer to store length as 2 words
			l*=8;	//length in bits
			uint32_t *p = (uint32_t*)&l;
			uint32_t *q = (uint32_t*)&out[i];
			*q = *(p+1);
			*(q+1) = *p;

			// The 64 bytes in the message block can now be used
			// to initialise the 64 4-byte words in the message schedule W[64]
			// REUSE i
			uint8_t *r = (uint8_t*)M;
			uint8_t *s = (uint8_t*)W;
			for(i=0;i<64;i++) s[i] = r[i];
			for(i=16;i<64;i++) W[i] = SIG1(W[i-2]) + W[i-7] + SIG0(W[i-15]) + W[i-16];

			// set initial hash values
			initHash(H);

			// Now calc the hash
			sha256_transform(W,H);

			// update the counters
			reduction_idx += 1;
			count -= 1;

		} // while(count>0)

		// copy comp_hash to final hash
		for(i=0;i<8;i++) data->final_hash[i] = H[i];
		data->sublinks = thread_idx;

		__syncthreads();
	} // if(thread_idx<LINKS)
} // kernel

//=================================Main Code==================================

int main(int argc, char **argv) {

	const char *tables_path = "./rbt/RbowTab_tables_0.rbt";
	char rbt_file[128];
	FILE *fp_rbow, *fp_tables;
	TableHeader *header, *dev_header;
	TableEntry *entry, *dev_entry, *target, *check, *compare;
	TableHeader *subchain_header;
	TableEntry *subchain_entry;
	int i,di,dx;
	int solutions, collisions;
	
	printf("searchtable_v4 (cuda).\n");
	printf("Search a merged Rainbow Table for a selected password.\n");
	
	// Sanity checks. In this case assert (LINKS % THREADS)==0
	if((LINKS%THREADS)!=0) {
		printf("Sanity test in csearch failed.\n");
		exit(1);
	}
	// calculate number of blocks to launch
	const int threads=THREADS;				// threads per block
	const int blocks = (LINKS+THREADS-1)/threads;		// number of thread blocks

	target = (TableEntry*)malloc(sizeof(TableEntry));

#if(1)	
	// get test data - this is a known password/hash pair	
	srand(time(NULL));
	fp_rbow = fopen("./rbt/RbowTab_merge.rbt","r");
	get_rnd_table_entry(target, fp_rbow);
	fclose(fp_rbow);
	//Confirming selected target data.
	//Password: VI21tdV
	//Hash: 1b69ba30 d4c58d76 51d114fe c9f57390 73b5f0d4 84ee9870 f483f478 bce85866 
#else
	// setup known solution in 'target' for debug
	// associated hash is loaded into "input_hash"
	target->sublinks=0;
	strcpy(target->initial_password, "ZZ90syK");
	hash2uint32("b545a2399c2cb7dabf7ce8eae859789574266163d18a7296f715c34affca2b6f",
				(uint32_t*)&target->final_hash[0]);
#endif
	
	// confirmation	of target
	printf("\nConfirming selected target data.\nPassword: %s\nHash: ", target->initial_password);
	for(dx=0;dx<8;dx++) printf("%08x ", target->final_hash[dx]);
	printf("\n");
	

	// allocate space for subchain tables
	subchain_header = (TableHeader*)malloc(sizeof(TableHeader));
	subchain_entry  = (TableEntry*)malloc(sizeof(TableEntry)*LINKS);
	if((subchain_header==NULL)||(subchain_entry==NULL)) {
		printf("Error - searchtable: Subchain host memory allocation failed.\n");
		exit(1);
	}

	// set up the subchain table
	// subchain_header->hdr_size = sizeof(TableHeader);
	// subchain_header->f1 = 0x00000000U;
	for(i=0;i<LINKS;i++) {
		(subchain_entry+i)->sublinks=0;
		for(di=0;di<8;di++) {
			(subchain_entry+i)->input_hash[di] = target->final_hash[di];
			(subchain_entry+i)->final_hash[di] = 0xffffffff;
		}
	}

	// allocate device memory
	HANDLE_ERROR(hipMalloc((void**)&dev_header,sizeof(TableHeader)));
	HANDLE_ERROR(hipMalloc((void**)&dev_entry,sizeof(TableEntry)*LINKS));

	// Copy entries to device
	HANDLE_ERROR(hipMemcpy(dev_entry, subchain_entry, sizeof(TableEntry)*LINKS, hipMemcpyHostToDevice));

	// launch kernel
	printf("Launching %d blocks of %d threads\n",blocks,threads);
	kernel<<<blocks,threads>>>(dev_header,dev_entry);

	// copy entries to host
	HANDLE_ERROR(hipMemcpy(subchain_entry, dev_entry, sizeof(TableEntry)*LINKS, hipMemcpyDeviceToHost));

	// Search Rainbow Tables for solution
	
	// ----------set up the Rainbow table----------
	fp_tables = fopen(tables_path,"r");
	if(fp_tables==NULL) {
		printf("Error - unable to open %s\n",tables_path);
		exit(1);
	}
	printf("Now looking for a valid solution\n");
	// look for a valid solution
	while((fscanf(fp_tables,"%s",rbt_file)) != EOF) {
		fp_rbow = fopen(rbt_file,"r");
		if(fp_rbow==NULL) {
			printf("Error - unable to open %s\n",rbt_file);
			exit(1);
		} else {
			printf("\nUsing table %s\n", rbt_file);
		}
		header = (TableHeader*)malloc(sizeof(TableHeader));
		fread(header,sizeof(TableHeader),1,fp_rbow);
		entry = (TableEntry*)malloc(sizeof(TableEntry)*header->entries);
		fread(entry,sizeof(TableEntry),header->entries,fp_rbow);
		fclose(fp_rbow);
		
		// try to match a subchain final_hash against final_hash in main table
		// if match found - report chain_index and link_index.
		printf("Looking for a matching chain...\n");
		collisions=0;
		solutions=0;
		check = (TableEntry*)malloc(sizeof(TableEntry)*(LINKS));
		for(i=0;i<LINKS;i++) {				
			// left points to candidate
			// left = (subchain_entry+i)->final_hash;
			// right points to merged table ordered by ascending final_hash
			// right = (entry+di)->final_hash;
			/*
			 * if compare == 1,  candidate > merged, continue
			 * if compare == -1, candidate < merged, break
			 */			
			compare = (TableEntry*)bsearch((subchain_entry+i), entry,
				header->entries, sizeof(TableEntry), hash_compare_32bit );

			if(compare!=NULL) {
				// printf("?");
				// Forward calculate the chain (entry+di) to (possibly) recover 
				// the password/hash pair.
				// check = (TableEntry*)malloc(sizeof(TableEntry)*(i+1));
				strcpy(check->initial_password,compare->initial_password);							
				compute_chain(check,i+1);			
				if(hash_compare_uint32_t((target)->final_hash,(check+i)->final_hash)==0) {
					printf("\033[31m");
					printf("\n=====SOLUTION FOUND===== \n%s\n",(check+i)->initial_password);
					for(dx=0;dx<8;dx++) printf("%08x ", (target)->final_hash[dx]);
					printf("\033[0m");
					solutions++;
					free(check);
					free(entry);
					free(header);
					goto found;
				} else { 
					//printf("- ");
					collisions++; 
				}
				//free(check);				 
			} else { 
				//printf(". "); 
			} // if (compare)				
		} // for[i=0]
		free(check);		
		free(entry);
		free(header);
	} // while !EOF
	found:
	// next two free() moved outside loop
		free(subchain_header);
		free(subchain_entry);
	// end move
	printf("\nThis run had %d collisions.\n",collisions);
	// free memory and file handles 
	fclose(fp_tables);
	free(target);
	return(0);
}


