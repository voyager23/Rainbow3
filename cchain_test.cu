#include "hip/hip_runtime.h"
/*	
	*
	* cchain_test.cu
	* testbed code to investigate compute_chain
	*
*/

#ifndef __CUDA__
	#define __CUDA__
#endif

//===========================Include code======================================

#include "rainbow.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <endian.h>
#include <time.h>

//=========================Declarations=================================
#ifdef __CUDA__
__global__
#endif
void kernel(TableHeader *header, TableEntry *entry);

// Hash constants
#ifdef __CUDA__
__constant__
#endif
uint32_t k[64] = {
	   0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	   0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	   0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	   0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	   0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	   0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	   0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	   0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2 };
	   
//=========================Include Device Code==========================

#include "freduce.cu"
#include "initHash.cu"
#include "sha256_txfm.cu"
#include "utils.cu"

#define NODEBUG
void hash2uint32(char *hash_str, uint32_t *H) {
	// hash_str must be 64 byte hexadecimal string
	const int words=8;
	char buffer[9], *source=hash_str;
	int i,len;
	len = strlen(hash_str);
	if(len != sizeof(unsigned)*words*2) {
		printf("Error - hash2uint32: hash_str length=%d\n",len);
		exit(1);
	}	
	for(i=0;i<words;i++) {
		strncpy(buffer,source,8);
		buffer[8]='\0';
		sscanf(buffer,"%x",H+i);
		source+=8;
	}
#ifdef DEBUG
	printf("hash2uint32\nHash:%s\nH[8]:",hash_str);
	for(i=0;i<8;i++) printf("%08x",H[i]); 
	printf("\n");
	#undef DEBUG
#endif
}



//=================================Main Code==================================

int main(int argc, char **argv) {

	TableEntry *check;
	
	//void compute_chain(TableEntry *entry, int links)
	check = (TableEntry*)malloc(sizeof(TableEntry)*(LINKS));
	strcpy(check->initial_password, "MM49jhM");
	compute_chain(check,5);
	show_table_entries(check,0,5);
	free(check);
	return(0);
}


